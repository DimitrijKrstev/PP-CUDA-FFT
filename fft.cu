#include "hip/hip_runtime.h"
#include "fft.h"

__device__ uint32_t reverse_bits_gpu(uint32_t x, int logN) {
    return __brev(x) >> (32 - logN);
}

__host__ void precompute_twiddle_factors(hipDoubleComplex* twiddle, uint32_t N) {
    for (uint32_t j = 0; j < N / 2; j++) {
        double angle = -2.0 * M_PI * j / N;
        twiddle[j] = make_hipDoubleComplex(cos(angle), sin(angle));
    }
}

// FFT kernel
__global__ void fft_kernel(hipDoubleComplex* Y, hipDoubleComplex* twiddle, uint32_t N, int logN) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= N / 2) return;

    uint32_t rev = reverse_bits_gpu(i, logN);
    hipDoubleComplex temp = Y[i];
    Y[i] = Y[rev];
    Y[rev] = temp;

    __syncthreads();

    for (int s = 1; s <= logN; s++) {
        int mh = 1 << (s - 1);
        int m = mh << 1;

        int group = i / mh;
        int group_start = group * m;
        int j = i % mh;

        hipDoubleComplex a = Y[group_start + j];
        hipDoubleComplex b = hipCmul(twiddle[j * (N / m)], Y[group_start + j + mh]);

        Y[group_start + j] = hipCadd(a, b);
        Y[group_start + j + mh] = hipCsub(a, b);

        __syncthreads();
    }
}

int fft_gpu(const hipDoubleComplex* x, hipDoubleComplex* Y, uint32_t N) {
    if (N & (N - 1)) {
        fprintf(stderr, "N=%u must be a power of 2.\n", N);
        return -1;
    }

    int logN = (int)log2f((float)N);

    hipDoubleComplex* x_dev;
    hipDoubleComplex* Y_dev;
    hipDoubleComplex* twiddle_dev;

    hipMalloc(&x_dev, sizeof(hipDoubleComplex) * N);
    hipMalloc(&Y_dev, sizeof(hipDoubleComplex) * N);
    hipMalloc(&twiddle_dev, sizeof(hipDoubleComplex) * (N / 2));

    hipMemcpy(x_dev, x, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);

    hipDoubleComplex* twiddle = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * (N / 2));
    precompute_twiddle_factors(twiddle, N);
    hipMemcpy(twiddle_dev, twiddle, sizeof(hipDoubleComplex) * (N / 2), hipMemcpyHostToDevice);
    free(twiddle);

    int block_size = 256;
    int grid_size = (N / 2 + block_size - 1) / block_size;

    hipMemcpy(Y_dev, x_dev, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToDevice);
    fft_kernel<<<grid_size, block_size>>>(Y_dev, twiddle_dev, N, logN);

    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    hipMemcpy(Y, Y_dev, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);

    hipFree(x_dev);
    hipFree(Y_dev);
    hipFree(twiddle_dev);

    return EXIT_SUCCESS;
}

int main() {
    hipDoubleComplex* gpu_output = (hipDoubleComplex*)malloc(testN * sizeof(hipDoubleComplex));

    printf("Running Parallel FFT...\n");
    fft_gpu(testInput, gpu_output, testN);
    print_complex_array("Parallel FFT Output", gpu_output, testN);

    free(gpu_output);

    return 0;
}
