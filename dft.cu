#include "hip/hip_runtime.h"
#include "fft.h"

int dft(const hipDoubleComplex* x, hipDoubleComplex* Y, uint32_t N) {
    for (size_t k = 0; k < N; k++) {
        hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
        double c = -2.0 * M_PI * k;

        for (size_t n = 0; n < N; n++) {
            double a = (c * n) / N;
            double sin_a, cos_a;

            sincos(a, &sin_a, &cos_a);
            hipDoubleComplex twiddle = make_hipDoubleComplex(cos_a, -sin_a);
            sum = hipCadd(sum, hipCmul(x[n], twiddle));
        }
        Y[k] = sum;
    }
    return EXIT_SUCCESS;
}

int main() {
  hipDoubleComplex output[testN];
  dft(testInput, output, testN);

  print_complex_array("Standard dft output", output, testN);
}
