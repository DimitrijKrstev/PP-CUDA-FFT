#include "hip/hip_runtime.h"
#include "fft.h"

uint32_t reverse_bits(uint32_t x) {
    x = ((x & 0xAAAAAAAA) >> 1) | ((x & 0x55555555) << 1);
    x = ((x & 0xCCCCCCCC) >> 2) | ((x & 0x33333333) << 2);
    x = ((x & 0xF0F0F0F0) >> 4) | ((x & 0x0F0F0F0F) << 4);
    x = ((x & 0xFF00FF00) >> 8) | ((x & 0x00FF00FF) << 8);
    return (x >> 16) | (x << 16);
}

int fft(const hipDoubleComplex* x, hipDoubleComplex* Y, uint32_t N) {
    int logN = (int)log2f((float)N);

    for (uint32_t i = 0; i < N; i++) {
        uint32_t rev = reverse_bits(i) >> (32 - logN);
        Y[i] = x[rev];
    }

    for (int s = 1; s <= logN; s++) {
        int m = 1 << s;      // Current transform size
        int mh = m >> 1;     // Half of the current transform size

        // Precompute twiddle factor for this stage
        float angle = -2.0f * M_PI / m;
        hipDoubleComplex twiddle = make_hipDoubleComplex(cosf(angle), sinf(angle));

        for (uint32_t k = 0; k < N; k += m) {
            hipDoubleComplex twiddle_factor = make_hipDoubleComplex(1.0f, 0.0f);

            for (int j = 0; j < mh; j++) {
                hipDoubleComplex a = Y[k + j];
                hipDoubleComplex b = hipCmul(twiddle_factor, Y[k + j + mh]);

                // Update Y[k + j] and Y[k + j + mh]
                Y[k + j] = hipCadd(a, b);
                Y[k + j + mh] = hipCsub(a, b);

                // Update twiddle factor
                twiddle_factor = hipCmul(twiddle_factor, twiddle);
            }
        }
    }

    return EXIT_SUCCESS;
}

int main(){
  hipDoubleComplex output[testN];
  fft(testInput, output, testN);

  print_complex_array("FFT-iterative output", output, testN);
}
